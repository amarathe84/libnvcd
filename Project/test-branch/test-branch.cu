#include "hip/hip_runtime.h"
/* This program uses the host CURAND API to generate 100
 * pseudorandom floats.
 */

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <nvcd/nvcd.cuh>

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
      printf("Error at %s:%d\n",__FILE__,__LINE__);            \
return EXIT_FAILURE;}} while(0)

__global__ void branch_kernel_bad(int* iterations) {
  
}

int main() {
  size_t n = 100;
  size_t i;

  hiprandGenerator_t gen;
  unsigned int *devData, *hostData;
  
  /* Allocate n ints on host */
  hostData = (unsigned int *)calloc(n, sizeof(unsigned int));

  /* Allocate n ints on device */
  CUDA_RUNTIME_FN(hipMalloc((void **)&devData, n * sizeof(unsigned int)));

  /* Create pseudo-random number generator */
  CURAND_CALL(hiprandCreateGenerator(&gen,
                                    HIPRAND_RNG_PSEUDO_DEFAULT));

  /* Set seed */
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

  /* Generate n ints on device */
  CURAND_CALL(hiprandGenerate(gen, devData, n));

  /* Copy device memory to host */
  CUDA_RUNTIME_FN(hipMemcpy(hostData,
                       devData,
                       n * sizeof(unsigned int),
                       hipMemcpyDeviceToHost));

  /* Set seed */
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

  /* Generate n floats on device */
  CURAND_CALL(hiprandGenerate(gen,
                             devData,
                             n));

  /* Copy device memory to host */
  CUDA_RUNTIME_FN(hipMemcpy(hostData,
                       devData,
                       n * sizeof(unsigned int),
                       hipMemcpyDeviceToHost));
  
  /* Show result */
  for(i = 0; i < n; i++) {
    printf("Value [%" PRIu64  "]: %" PRIu32 "\n", i, hostData[i]);
  }

  printf("%s", "\n");

  /* Cleanup */
  CURAND_CALL(hiprandDestroyGenerator(gen));
  CUDA_RUNTIME_FN(hipFree(devData));

  free(hostData);

  return EXIT_SUCCESS;
}
