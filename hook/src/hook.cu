
#include <hip/hip_runtime_api.h>

#define NVCD_HEADER_IMPL
#include <nvcd/nvcd.cuh>
#undef NVCD_HEADER_IMPL
// See comments for answer: https://stackoverflow.com/a/1777479
//#define __USE_GNU
#include <dlfcn.h>

template <class TKernFunType, class ...TArgs>
static inline hipError_t nvcd_run_metrics2(const TKernFunType& kernel, 				     
				     TArgs... args) {
  cupti_event_data_t* __e = nvcd_get_events();                           
  
  ASSERT(__e->is_root == true);                                       
  ASSERT(__e->initialized == true);                                   
  ASSERT(__e->metric_data != NULL);                                   
  ASSERT(__e->metric_data->initialized == true);                      

  hipError_t result = hipSuccess;
  for (uint32_t i = 0; result == hipSuccess && i < __e->metric_data->num_metrics; ++i) {      
    cupti_event_data_begin(&__e->metric_data->event_data[i]);         

    while (result == hipSuccess && !cupti_event_data_callback_finished(&__e->metric_data->event_data[i])) {
      kernel(args...);                       
      CUDA_RUNTIME_FN(hipDeviceSynchronize());                       
      g_run_info->run_kernel_count_inc();				
    }                                                                 
                                                                        
    cupti_event_data_end(&__e->metric_data->event_data[i]);
  }

  return result;
}


template <class TKernFunType, class ...TArgs>
static inline hipError_t nvcd_run2(const TKernFunType& kernel, 
				    TArgs... args) {
  
  cupti_event_data_begin(nvcd_get_events());
  hipError_t result = hipSuccess;
  while (result == hipSuccess && !nvcd_host_finished()) {                                     
    result = kernel(args...);                       
    CUDA_RUNTIME_FN(hipDeviceSynchronize());                         
    g_run_info->run_kernel_count_inc();			
  }                                                                   
  cupti_event_data_end(nvcd_get_events());    

  if (result == hipSuccess && g_nvcd.opt_calc_metrics) {  
    result = nvcd_run_metrics2(kernel, args...);
  }

  return result;
}

C_LINKAGE_START




typedef hipError_t (*cudaLaunchKernel_fn_t)(const void* func, dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream);

static cudaLaunchKernel_fn_t real_cudaLaunchKernel = NULL;

NVCD_EXPORT __host__ hipError_t hipLaunchKernel(reinterpret_cast<const void*>(const void* func),
						  dim3 gridDim,
						  dim3 blockDim,
						  void** args,
						  size_t sharedMem,
						  hipStream_t stream) {
  
  if (real_cudaLaunchKernel == NULL) {
    real_cudaLaunchKernel = (cudaLaunchKernel_fn_t) dlsym(RTLD_NEXT, "hipLaunchKernel");
  }
  printf("[HOOK %s]\n", __FUNC__);
  nvcd_host_begin(gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z);
  hipError_t ret = nvcd_run2(real_cudaLaunchKernel, func, gridDim, blockDim, args, sharedMem, stream);
  nvcd_host_end();
  return ret;
}



C_LINKAGE_END
